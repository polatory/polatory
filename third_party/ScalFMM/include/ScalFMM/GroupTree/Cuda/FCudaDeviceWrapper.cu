#include "hip/hip_runtime.h"
#include "FCudaDeviceWrapper.hpp"
#include "FCudaTreeCoordinate.hpp"
#include "FCudaStructParams.hpp"


#define FMGetOppositeNeighIndex(index) (27-(index)-1)
#define FMGetOppositeInterIndex(index) (343-(index)-1)

#define FCudaMax(x,y) ((x)<(y) ? (y) : (x))
#define FCudaMin(x,y) ((x)>(y) ? (y) : (x))


template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__global__ void FCuda__bottomPassPerform(unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
                                         unsigned char* containersPtr, std::size_t containersSize,
                                         CudaKernelClass* kernel){
    CellContainerClass leafCells(leafCellsPtr, leafCellsSize, leafCellsUpPtr, nullptr);
    ParticleContainerGroupClass containers(containersPtr, containersSize, nullptr);

    for(int leafIdx = blockIdx.x ; leafIdx < leafCells.getNumberOfCellsInBlock() ; leafIdx += gridDim.x){
        typename CellContainerClass::CompleteCellClass cell = leafCells.getUpCell(leafIdx);
        ParticleGroupClass particles = containers.template getLeaf<ParticleGroupClass>(leafIdx);
        FCudaAssertLF(leafCells.getCellMortonIndex(leafIdx) == containers.getLeafMortonIndex(leafIdx));
        kernel->P2M(cell, &particles);
    }
}

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__host__ void FCuda__bottomPassCallback(unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
                                        unsigned char* containersPtr, std::size_t containersSize,
                                        CudaKernelClass* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize){
    FCuda__bottomPassPerform
            <SymboleCellClass, PoleCellClass, LocalCellClass,
            CellContainerClass, ParticleContainerGroupClass, ParticleGroupClass, CudaKernelClass>
            <<<inGridSize, inBlocksSize, 0, currentStream>>>
                            (leafCellsPtr, leafCellsSize,leafCellsUpPtr,
                             containersPtr, containersSize,
                             kernel);
    FCudaCheckAfterCall();
    FCudaCheck(hipStreamSynchronize(currentStream));
}


/////////////////////////////////////////////////////////////////////////////////////
/// Upward Pass
/////////////////////////////////////////////////////////////////////////////////////

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__global__ void FCuda__upwardPassPerform(unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
                                         unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
                                         int idxLevel, CudaKernelClass* kernel){
    CellContainerClass currentCells(currentCellsPtr, currentCellsSize,currentCellsUpPtr,nullptr);
    CellContainerClass subCellGroup(childCellsPtr, childCellsSize,childCellsUpPtr,nullptr);

    const MortonIndex firstParent = FCudaMax(currentCells.getStartingIndex(), subCellGroup.getStartingIndex()>>3);
    const MortonIndex lastParent = FCudaMin(currentCells.getEndingIndex()-1, (subCellGroup.getEndingIndex()-1)>>3);

    int idxParentCell = currentCells.getCellIndex(firstParent);
    int idxChildCell = subCellGroup.getFistChildIdx(firstParent);

    while(true){
        typename CellContainerClass::CompleteCellClass cell = currentCells.getUpCell(idxParentCell);
        typename CellContainerClass::CompleteCellClass child[8];


        for(int idxChild = 0 ; idxChild < 8 ; ++idxChild){
            child[idxChild].symb = nullptr;
        }

        do{
            const int idxChild = ((subCellGroup.getCellMortonIndex(idxChildCell)) & 7);
            child[idxChild] = subCellGroup.getUpCell(idxChildCell);

            idxChildCell += 1;
        }while(idxChildCell != subCellGroup.getNumberOfCellsInBlock() && cell.symb->mortonIndex == (subCellGroup.getCellMortonIndex(idxChildCell)>>3));

        kernel->M2M(cell, child, idxLevel);

        if(currentCells.getCellMortonIndex(idxParentCell) == lastParent){
            break;
        }

        idxParentCell += 1;
    }
}

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__host__ void FCuda__upwardPassCallback(unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
                                        unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
                                        int idxLevel, CudaKernelClass* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize){

    FCuda__upwardPassPerform
            <SymboleCellClass, PoleCellClass, LocalCellClass,
            CellContainerClass, ParticleContainerGroupClass, ParticleGroupClass, CudaKernelClass>
            <<<inGridSize, inBlocksSize, 0, currentStream>>>
                            (currentCellsPtr, currentCellsSize,currentCellsUpPtr,
                             childCellsPtr, childCellsSize,childCellsUpPtr,
                             idxLevel, kernel);
    FCudaCheckAfterCall();
    FCudaCheck(hipStreamSynchronize(currentStream));
}



/////////////////////////////////////////////////////////////////////////////////////
/// Transfer Pass Mpi
/////////////////////////////////////////////////////////////////////////////////////
#ifdef SCALFMM_USE_MPI
template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__global__  void FCuda__transferInoutPassPerformMpi(unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
                                                  unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
                                                  const int* safeInteractions, int nbSafeInteractions, int idxLevel, const OutOfBlockInteraction* outsideInteractions,
                                                  int nbOutsideInteractions, CudaKernelClass* kernel){

    CellContainerClass currentCells(currentCellsPtr, currentCellsSize, nullptr, currentCellsDownPtr);
    CellContainerClass cellsOther(externalCellsPtr, externalCellsSize, externalCellsUpPtr, nullptr);

    for(int cellIdx = blockIdx.x ; cellIdx < nbSafeInteractions ; cellIdx += gridDim.x){
        for(int outInterIdx = safeInteractions[cellIdx] ; outInterIdx < safeInteractions[cellIdx+1] ; ++outInterIdx){
            const int cellPos = cellsOther.getCellIndex(outsideInteractions[outInterIdx].outIndex);
            if(cellPos != -1){
                typename CellContainerClass::CompleteCellClass interCell = cellsOther.getUpCell(cellPos);
                FCudaAssertLF(interCell.symb->mortonIndex == outsideInteractions[outInterIdx].outIndex);
                typename CellContainerClass::CompleteCellClass cell = currentCells.getDownCell(outsideInteractions[outInterIdx].insideIdxInBlock);
                FCudaAssertLF(cell.symb->mortonIndex == outsideInteractions[outInterIdx].insideIndex);

                kernel->M2L( cell , &interCell, &outsideInteractions[outInterIdx].relativeOutPosition, 1, idxLevel);
            }
        }
    }
}

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__host__ void FCuda__transferInoutPassCallbackMpi(unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
                                                  unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
                                                  int idxLevel, const OutOfBlockInteraction* outsideInteractions,
                                                  int nbOutsideInteractions, const int* safeInteractions, int nbSafeInteractions, CudaKernelClass* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize){
    OutOfBlockInteraction* cuOutsideInteractions;
    FCudaCheck( hipMalloc(&cuOutsideInteractions,nbOutsideInteractions*sizeof(OutOfBlockInteraction)) );
    FCudaCheck( hipMemcpy( cuOutsideInteractions, outsideInteractions, nbOutsideInteractions*sizeof(OutOfBlockInteraction),
                hipMemcpyHostToDevice ) );

    int* cuSafeInteractions;
    FCudaCheck( hipMalloc(&cuSafeInteractions,(nbSafeInteractions+1)*sizeof(int)) );
    FCudaCheck( hipMemcpy( cuSafeInteractions, safeInteractions, (nbSafeInteractions+1)*sizeof(int),
                hipMemcpyHostToDevice ) );

    FCuda__transferInoutPassPerformMpi
            <SymboleCellClass, PoleCellClass, LocalCellClass,
            CellContainerClass, ParticleContainerGroupClass, ParticleGroupClass, CudaKernelClass>
            <<<inGridSize, inBlocksSize, 0, currentStream>>>(currentCellsPtr, currentCellsSize, currentCellsDownPtr,
                                       externalCellsPtr, externalCellsSize, externalCellsUpPtr,
                                       cuSafeInteractions, nbSafeInteractions, idxLevel, cuOutsideInteractions, nbOutsideInteractions, kernel);
    FCudaCheckAfterCall();
    FCudaCheck(hipStreamSynchronize(currentStream));

    FCudaCheck(hipFree(cuSafeInteractions));
    FCudaCheck(hipFree(cuOutsideInteractions));
}
#endif

/////////////////////////////////////////////////////////////////////////////////////
/// Transfer Pass
/////////////////////////////////////////////////////////////////////////////////////


template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__global__  void FCuda__transferInPassPerform(unsigned char* currentCellsPtr, std::size_t currentCellsSize,
                                              unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
                                              int idxLevel, CudaKernelClass* kernel){

    CellContainerClass currentCells(currentCellsPtr, currentCellsSize, currentCellsUpPtr, currentCellsDownPtr);

    const MortonIndex blockStartIdx = currentCells.getStartingIndex();
    const MortonIndex blockEndIdx = currentCells.getEndingIndex();

    for(int cellIdx = blockIdx.x ; cellIdx < currentCells.getNumberOfCellsInBlock() ; cellIdx += gridDim.x){
        typename CellContainerClass::CompleteCellClass cell = currentCells.getDownCell(cellIdx);

        MortonIndex interactionsIndexes[189];
        int interactionsPosition[189];
        const int3 coord = (FCudaTreeCoordinate::ConvertCoordinate(cell.symb->coordinates));
        int counter = FCudaTreeCoordinate::GetInteractionNeighbors(coord, idxLevel,interactionsIndexes,interactionsPosition);

        typename CellContainerClass::CompleteCellClass interactions[189];
        int counterExistingCell = 0;

        for(int idxInter = 0 ; idxInter < counter ; ++idxInter){
            if( blockStartIdx <= interactionsIndexes[idxInter] && interactionsIndexes[idxInter] < blockEndIdx ){
                const int cellPos = currentCells.getCellIndex(interactionsIndexes[idxInter]);
                if(cellPos != -1){
                    typename CellContainerClass::CompleteCellClass interCell = currentCells.getUpCell(cellPos);
                    interactions[counterExistingCell] = interCell;
                    interactionsPosition[counterExistingCell] = interactionsPosition[idxInter];
                    counterExistingCell += 1;
                }
            }
        }

        kernel->M2L( cell , interactions, interactionsPosition, counterExistingCell, idxLevel);
    }
}

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__host__ void FCuda__transferInPassCallback(unsigned char* currentCellsPtr, std::size_t currentCellsSize,
                                            unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
                                            int idxLevel, CudaKernelClass* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize){

    FCuda__transferInPassPerform
            <SymboleCellClass, PoleCellClass, LocalCellClass,
            CellContainerClass, ParticleContainerGroupClass, ParticleGroupClass, CudaKernelClass>
            <<<inGridSize, inBlocksSize, 0, currentStream>>>(currentCellsPtr, currentCellsSize,
                                                                currentCellsUpPtr, currentCellsDownPtr,
                                                                idxLevel, kernel);
    FCudaCheckAfterCall();
    FCudaCheck(hipStreamSynchronize(currentStream));
}


template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__global__ void FCuda__transferInoutPassPerform(unsigned char* currentCellsPtr, std::size_t currentCellsSize,
                                                unsigned char* currentCellsDownPtr,
                                                unsigned char* externalCellsPtr, std::size_t externalCellsSize,
                                                unsigned char* externalCellsUpPtr,
                                                int idxLevel, int mode, const OutOfBlockInteraction* outsideInteractions,
                                                int nbOutsideInteractions,
                                                const int* safeInteractions, int nbSafeInteractions, CudaKernelClass* kernel){

    CellContainerClass currentCells(currentCellsPtr, currentCellsSize, nullptr, currentCellsDownPtr);
    CellContainerClass cellsOther(externalCellsPtr, externalCellsSize, externalCellsUpPtr, nullptr);

    if(mode == 1){
        for(int cellIdx = blockIdx.x ; cellIdx < nbSafeInteractions ; cellIdx += gridDim.x){
            for(int outInterIdx = safeInteractions[cellIdx] ; outInterIdx < safeInteractions[cellIdx+1] ; ++outInterIdx){
                typename CellContainerClass::CompleteCellClass interCell = cellsOther.getUpCell(outsideInteractions[outInterIdx].outsideIdxInBlock);
                FCudaAssertLF(interCell.symb->mortonIndex == outsideInteractions[outInterIdx].outIndex);
                typename CellContainerClass::CompleteCellClass cell = currentCells.getDownCell(outsideInteractions[outInterIdx].insideIdxInBlock);
                FCudaAssertLF(cell.symb->mortonIndex == outsideInteractions[outInterIdx].insideIndex);

                kernel->M2L( cell , &interCell, &outsideInteractions[outInterIdx].relativeOutPosition, 1, idxLevel);
            }
        }
    }
    else{
        for(int cellIdx = blockIdx.x ; cellIdx < nbSafeInteractions ; cellIdx += gridDim.x){
            for(int outInterIdx = safeInteractions[cellIdx] ; outInterIdx < safeInteractions[cellIdx+1] ; ++outInterIdx){
                typename CellContainerClass::CompleteCellClass cell = cellsOther.getUpCell(outsideInteractions[outInterIdx].insideIdxInBlock);
                FCudaAssertLF(cell.symb->mortonIndex == outsideInteractions[outInterIdx].insideIndex);
                typename CellContainerClass::CompleteCellClass interCell = currentCells.getDownCell(outsideInteractions[outInterIdx].outsideIdxInBlock);
                FCudaAssertLF(interCell.symb->mortonIndex == outsideInteractions[outInterIdx].outIndex);

                const int otherPosition = FMGetOppositeInterIndex(outsideInteractions[outInterIdx].relativeOutPosition);
                kernel->M2L( interCell , &cell, &otherPosition, 1, idxLevel);
            }
        }
    }
}


template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__host__ void FCuda__transferInoutPassCallback(unsigned char* currentCellsPtr, std::size_t currentCellsSize,
                                               unsigned char* currentCellsDownPtr,
                                               unsigned char* externalCellsPtr, std::size_t externalCellsSize,
                                               unsigned char* externalCellsUpPtr,
                                               int idxLevel, int mode,
                                               const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
                                               const int* safeInteractions, int nbSafeInteractions,
                                               CudaKernelClass* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize){
    OutOfBlockInteraction* cuOutsideInteractions;
    FCudaCheck( hipMalloc(&cuOutsideInteractions,nbOutsideInteractions*sizeof(OutOfBlockInteraction)) );
    FCudaCheck( hipMemcpy( cuOutsideInteractions, outsideInteractions, nbOutsideInteractions*sizeof(OutOfBlockInteraction),
                hipMemcpyHostToDevice ) );

    int* cuSafeInteractions;
    FCudaCheck( hipMalloc(&cuSafeInteractions,(nbSafeInteractions+1)*sizeof(int)) );
    FCudaCheck( hipMemcpy( cuSafeInteractions, safeInteractions, (nbSafeInteractions+1)*sizeof(int),
                hipMemcpyHostToDevice ) );

    FCuda__transferInoutPassPerform
            <SymboleCellClass, PoleCellClass, LocalCellClass,
            CellContainerClass, ParticleContainerGroupClass, ParticleGroupClass, CudaKernelClass>
            <<<inGridSize, inBlocksSize, 0, currentStream>>>(currentCellsPtr, currentCellsSize,
                                                                currentCellsDownPtr,
                                                                externalCellsPtr, externalCellsSize,
                                                                externalCellsUpPtr,
                                                                idxLevel, mode,
                                                                cuOutsideInteractions, nbOutsideInteractions,
                                                                cuSafeInteractions, nbSafeInteractions,
                                                                kernel);
    FCudaCheckAfterCall();
    FCudaCheck(hipStreamSynchronize(currentStream));

    FCudaCheck(hipFree(cuOutsideInteractions));
    FCudaCheck(hipFree(cuSafeInteractions));
}


/////////////////////////////////////////////////////////////////////////////////////
/// Downard Pass
/////////////////////////////////////////////////////////////////////////////////////

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__global__ void FCuda__downardPassPerform(unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
                                          unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
                                          int idxLevel, CudaKernelClass* kernel){
    CellContainerClass currentCells(currentCellsPtr, currentCellsSize,nullptr,currentCellsDownPtr);
    CellContainerClass subCellGroup(childCellsPtr, childCellsSize,nullptr,childCellsDownPtr);

    const MortonIndex firstParent = FCudaMax(currentCells.getStartingIndex(), subCellGroup.getStartingIndex()>>3);
    const MortonIndex lastParent = FCudaMin(currentCells.getEndingIndex()-1, (subCellGroup.getEndingIndex()-1)>>3);

    int idxParentCell = currentCells.getCellIndex(firstParent);
    int idxChildCell = subCellGroup.getFistChildIdx(firstParent);

    while(true){
        typename CellContainerClass::CompleteCellClass cell = currentCells.getDownCell(idxParentCell);
        typename CellContainerClass::CompleteCellClass child[8];


        for(int idxChild = 0 ; idxChild < 8 ; ++idxChild){
            child[idxChild].symb = nullptr;
        }

        do{
            const int idxChild = ((subCellGroup.getCellMortonIndex(idxChildCell)) & 7);
            child[idxChild] = subCellGroup.getDownCell(idxChildCell);

            idxChildCell += 1;
        }while(idxChildCell != subCellGroup.getNumberOfCellsInBlock() && cell.symb->mortonIndex == (subCellGroup.getCellMortonIndex(idxChildCell)>>3));

        kernel->L2L(cell, child, idxLevel);

        if(currentCells.getCellMortonIndex(idxParentCell) == lastParent){
            break;
        }

        idxParentCell += 1;
    }
}

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__host__ void FCuda__downardPassCallback(unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
                                        unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
                                         int idxLevel, CudaKernelClass* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize){

    FCuda__downardPassPerform
            <SymboleCellClass, PoleCellClass, LocalCellClass,
            CellContainerClass, ParticleContainerGroupClass, ParticleGroupClass, CudaKernelClass>
            <<<inGridSize, inBlocksSize, 0, currentStream>>>
            (currentCellsPtr, currentCellsSize, currentCellsDownPtr, childCellsPtr, childCellsSize, childCellsDownPtr,
             idxLevel, kernel);
    FCudaCheckAfterCall();
    FCudaCheck(hipStreamSynchronize(currentStream));
}

/////////////////////////////////////////////////////////////////////////////////////
/// Direct Pass MPI
/////////////////////////////////////////////////////////////////////////////////////
#ifdef SCALFMM_USE_MPI
template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__global__ void FCuda__directInoutPassPerformMpi(unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
                                                 unsigned char* externalContainersPtr, std::size_t externalContainersSize,
                                                 const OutOfBlockInteraction* outsideInteractions,
                                                 int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
                                                 const int treeHeight, CudaKernelClass* kernel){

    ParticleContainerGroupClass containers(containersPtr, containersSize, containersDownPtr);
    ParticleContainerGroupClass containersOther(externalContainersPtr, externalContainersSize, nullptr);

    for(int leafIdx = blockIdx.x ; leafIdx < counterOuterCell ; leafIdx += gridDim.x){
        for(int outInterIdx = safeOuterInteractions[leafIdx] ; outInterIdx < safeOuterInteractions[leafIdx+1] ; ++outInterIdx){
            const int leafPos = containersOther.getLeafIndex(outsideInteractions[outInterIdx].outIndex);
            if(leafPos != -1){
                ParticleGroupClass interParticles = containersOther.template getLeaf<ParticleGroupClass>(leafPos);
                ParticleGroupClass particles = containers.template getLeaf<ParticleGroupClass>(outsideInteractions[outInterIdx].insideIdxInBlock);

                kernel->P2PRemote( FCudaTreeCoordinate::GetPositionFromMorton(outsideInteractions[outInterIdx].insideIndex, treeHeight-1),
                                   &particles, &particles , &interParticles, &outsideInteractions[outInterIdx].relativeOutPosition, 1);
            }
        }
    }
}

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__host__ void FCuda__directInoutPassCallbackMpi(unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
                                                unsigned char* externalContainersPtr, std::size_t externalContainersSize,
                                                const OutOfBlockInteraction* outsideInteractions,
                                                int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
                                                const int treeHeight, CudaKernelClass* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize){
    OutOfBlockInteraction* cuOutsideInteractions;
    FCudaCheck( hipMalloc(&cuOutsideInteractions,nbOutsideInteractions*sizeof(OutOfBlockInteraction)) );
    FCudaCheck( hipMemcpy( cuOutsideInteractions, outsideInteractions, nbOutsideInteractions*sizeof(OutOfBlockInteraction),
                hipMemcpyHostToDevice ) );

    int* cuSafeOuterInteractions;
    FCudaCheck( hipMalloc(&cuSafeOuterInteractions,(counterOuterCell+1)*sizeof(int)) );
    FCudaCheck( hipMemcpy( cuSafeOuterInteractions, safeOuterInteractions, (counterOuterCell+1)*sizeof(int),
                hipMemcpyHostToDevice ) );

    FCuda__directInoutPassPerformMpi
            <SymboleCellClass, PoleCellClass, LocalCellClass,
            CellContainerClass, ParticleContainerGroupClass, ParticleGroupClass, CudaKernelClass>
            <<<inGridSize, inBlocksSize, 0, currentStream>>>(containersPtr, containersSize, containersDownPtr,
                                  externalContainersPtr, externalContainersSize,
                                  cuOutsideInteractions, nbOutsideInteractions, cuSafeOuterInteractions, counterOuterCell,
                                                             treeHeight, kernel);

    FCudaCheckAfterCall();
    FCudaCheck(hipStreamSynchronize(currentStream));

    FCudaCheck(hipFree(cuOutsideInteractions));
    FCudaCheck(hipFree(cuSafeOuterInteractions));
}
#endif
/////////////////////////////////////////////////////////////////////////////////////
/// Direct Pass
/////////////////////////////////////////////////////////////////////////////////////


template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__global__ void FCuda__directInPassPerform(unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
                                           const int treeHeight, CudaKernelClass* kernel){

    ParticleContainerGroupClass containers(containersPtr, containersSize, containersDownPtr);

    const MortonIndex blockStartIdx = containers.getStartingIndex();
    const MortonIndex blockEndIdx = containers.getEndingIndex();

    for(int leafIdx = blockIdx.x ; leafIdx < containers.getNumberOfLeavesInBlock() ; leafIdx += gridDim.x){
        ParticleGroupClass particles = containers.template getLeaf<ParticleGroupClass>(leafIdx);
        const MortonIndex mindex = containers.getLeafMortonIndex(leafIdx);
        MortonIndex interactionsIndexes[26];
        int interactionsPosition[26];
        const int3 coord = FCudaTreeCoordinate::GetPositionFromMorton(mindex, treeHeight-1);
        int counter = FCudaTreeCoordinate::GetNeighborsIndexes(coord, treeHeight,interactionsIndexes,interactionsPosition);

        ParticleGroupClass interactionsObjects[26];
        int counterExistingCell = 0;

        for(int idxInter = 0 ; idxInter < counter ; ++idxInter){
            if( blockStartIdx <= interactionsIndexes[idxInter] && interactionsIndexes[idxInter] < blockEndIdx ){
                const int leafPos = containers.getLeafIndex(interactionsIndexes[idxInter]);
                if(leafPos != -1){
                    interactionsObjects[counterExistingCell] = containers.template getLeaf<ParticleGroupClass>(leafPos);
                    interactionsPosition[counterExistingCell] = interactionsPosition[idxInter];
                    counterExistingCell += 1;
                }
            }
        }

        kernel->P2P( coord, &particles, &particles , interactionsObjects, interactionsPosition, counterExistingCell);
    }
}

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__host__ void FCuda__directInPassCallback(unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
                                          const int treeHeight, CudaKernelClass* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize){
    FCuda__directInPassPerform
            <SymboleCellClass, PoleCellClass, LocalCellClass,
            CellContainerClass, ParticleContainerGroupClass, ParticleGroupClass, CudaKernelClass>
            <<<inGridSize, inBlocksSize, 0, currentStream>>>(containersPtr, containersSize, containersDownPtr,
                               treeHeight, kernel);
    FCudaCheckAfterCall();
    FCudaCheck(hipStreamSynchronize(currentStream));
}

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__global__ void FCuda__directInoutPassPerform(unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
                                              unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
                                              const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
                                              const int     safeOuterInteractions[], const int counterOuterCell,
                                              const OutOfBlockInteraction* insideInteractions,
                                              const int     safeInnterInteractions[], const int counterInnerCell,
                                              const int treeHeight, CudaKernelClass* kernel){

    ParticleContainerGroupClass containers(containersPtr, containersSize, containersDownPtr);
    ParticleContainerGroupClass containersOther(externalContainersPtr, externalContainersSize, externalContainersDownPtr);

    for(int leafIdx = blockIdx.x ; leafIdx < counterOuterCell ; leafIdx += gridDim.x){
        for(int outInterIdx = safeOuterInteractions[leafIdx] ; outInterIdx < safeOuterInteractions[leafIdx+1] ; ++outInterIdx){
            ParticleGroupClass interParticles = containersOther.template getLeaf<ParticleGroupClass>(outsideInteractions[outInterIdx].outsideIdxInBlock);
            ParticleGroupClass particles = containers.template getLeaf<ParticleGroupClass>(outsideInteractions[outInterIdx].insideIdxInBlock);

            FCudaAssertLF(containersOther.getLeafMortonIndex(outsideInteractions[outInterIdx].outsideIdxInBlock) == outsideInteractions[outInterIdx].outIndex);
            FCudaAssertLF(containers.getLeafMortonIndex(outsideInteractions[outInterIdx].insideIdxInBlock) == outsideInteractions[outInterIdx].insideIndex);

            kernel->P2POuter( FCudaTreeCoordinate::GetPositionFromMorton(outsideInteractions[outInterIdx].insideIndex, treeHeight-1),
                               &particles , &interParticles, &outsideInteractions[outInterIdx].relativeOutPosition, 1);
        }
    }

    for(int leafIdx = blockIdx.x ; leafIdx < counterInnerCell ; leafIdx += gridDim.x){
        for(int outInterIdx = safeInnterInteractions[leafIdx] ; outInterIdx < safeInnterInteractions[leafIdx+1] ; ++outInterIdx){

            ParticleGroupClass interParticles = containersOther.template getLeaf<ParticleGroupClass>(insideInteractions[outInterIdx].outsideIdxInBlock);
            ParticleGroupClass particles = containers.template getLeaf<ParticleGroupClass>(insideInteractions[outInterIdx].insideIdxInBlock);

            FCudaAssertLF(containersOther.getLeafMortonIndex(insideInteractions[outInterIdx].outsideIdxInBlock) == insideInteractions[outInterIdx].outIndex);
            FCudaAssertLF(containers.getLeafMortonIndex(insideInteractions[outInterIdx].insideIdxInBlock) == insideInteractions[outInterIdx].insideIndex);

            const int otherPosition = FMGetOppositeNeighIndex(insideInteractions[outInterIdx].relativeOutPosition);
            kernel->P2POuter( FCudaTreeCoordinate::GetPositionFromMorton(insideInteractions[outInterIdx].outIndex, treeHeight-1),
                               &interParticles , &particles, &otherPosition, 1);
        }
    }
}

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__host__ void FCuda__directInoutPassCallback(unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
                                             unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
                                             const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
                                             const int     safeOuterInteractions[], const int counterOuterCell,
                                                 const OutOfBlockInteraction* insideInteractions,
                                                 const int     safeInnterInteractions[], const int counterInnerCell,
                                             const int treeHeight, CudaKernelClass* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize){
    OutOfBlockInteraction* cuOutsideInteractions;
    FCudaCheck( hipMalloc(&cuOutsideInteractions,nbOutsideInteractions*sizeof(OutOfBlockInteraction)) );
    FCudaCheck( hipMemcpy( cuOutsideInteractions, outsideInteractions, nbOutsideInteractions*sizeof(OutOfBlockInteraction),
                hipMemcpyHostToDevice ) );

    OutOfBlockInteraction* cuInsideInteractions;
    FCudaCheck( hipMalloc(&cuInsideInteractions,nbOutsideInteractions*sizeof(OutOfBlockInteraction)) );
    FCudaCheck( hipMemcpy( cuInsideInteractions, insideInteractions, nbOutsideInteractions*sizeof(OutOfBlockInteraction),
                hipMemcpyHostToDevice ) );

    int* cuSafeOuterInteractions;
    FCudaCheck( hipMalloc(&cuSafeOuterInteractions,(counterOuterCell+1)*sizeof(int)) );
    FCudaCheck( hipMemcpy( cuSafeOuterInteractions, safeOuterInteractions, (counterOuterCell+1)*sizeof(int),
                hipMemcpyHostToDevice ) );

    int* cuSafeInnterInteractions;
    FCudaCheck( hipMalloc(&cuSafeInnterInteractions,(counterInnerCell+1)*sizeof(int)) );
    FCudaCheck( hipMemcpy( cuSafeInnterInteractions, safeInnterInteractions, (counterInnerCell+1)*sizeof(int),
                hipMemcpyHostToDevice ) );

    FCuda__directInoutPassPerform
            <SymboleCellClass, PoleCellClass, LocalCellClass,
            CellContainerClass, ParticleContainerGroupClass, ParticleGroupClass, CudaKernelClass>
            <<<inGridSize, inBlocksSize, 0, currentStream>>>(containersPtr, containersSize,containersDownPtr,
                                  externalContainersPtr, externalContainersSize,externalContainersDownPtr,
                                  cuOutsideInteractions, nbOutsideInteractions,
                                 cuSafeOuterInteractions,counterOuterCell,
                                  cuInsideInteractions,
                                  cuSafeInnterInteractions , counterInnerCell,
                                  treeHeight, kernel);

    FCudaCheckAfterCall();
    FCudaCheck(hipStreamSynchronize(currentStream));

    FCudaCheck(hipFree(cuOutsideInteractions));
    FCudaCheck(hipFree(cuInsideInteractions));
    FCudaCheck(hipFree(cuSafeOuterInteractions));
    FCudaCheck(hipFree(cuSafeInnterInteractions));
}


/////////////////////////////////////////////////////////////////////////////////////
/// Merge Pass
/////////////////////////////////////////////////////////////////////////////////////


template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__global__ void FCuda__mergePassPerform(unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
                                        unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
                                        CudaKernelClass* kernel){    
    CellContainerClass leafCells(leafCellsPtr,leafCellsSize, nullptr, leafCellsDownPtr);
    ParticleContainerGroupClass containers(containersPtr,containersSize, containersDownPtr);

    for(int cellIdx = blockIdx.x ; cellIdx < leafCells.getNumberOfCellsInBlock() ; cellIdx += gridDim.x){
        typename CellContainerClass::CompleteCellClass cell = leafCells.getDownCell(cellIdx);
        FCudaAssertLF(cell.symb->mortonIndex == leafCells.getCellMortonIndex(cellIdx));
        ParticleGroupClass particles = containers.template getLeaf<ParticleGroupClass>(cellIdx);
        FCudaAssertLF(leafCells.getCellMortonIndex(cellIdx) == containers.getLeafMortonIndex(cellIdx));
        kernel->L2P(cell, &particles);
    }
}

template <class SymboleCellClass, class PoleCellClass, class LocalCellClass,
          class CellContainerClass, class ParticleContainerGroupClass, class ParticleGroupClass, class CudaKernelClass>
__host__ void FCuda__mergePassCallback(unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
                                       unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
                                       CudaKernelClass* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize){
    FCuda__mergePassPerform
            <SymboleCellClass, PoleCellClass, LocalCellClass,
            CellContainerClass, ParticleContainerGroupClass, ParticleGroupClass, CudaKernelClass>
            <<<inGridSize, inBlocksSize, 0, currentStream>>>(leafCellsPtr, leafCellsSize,leafCellsDownPtr,
                            containersPtr, containersSize,containersDownPtr,
                            kernel);

    FCudaCheckAfterCall();
    FCudaCheck(hipStreamSynchronize(currentStream));
}


template <class CudaKernelClass>
CudaKernelClass* FCuda__BuildCudaKernel(void* kernel){
    return CudaKernelClass::InitKernelKernel(kernel);
}

template <class CudaKernelClass>
void FCuda__ReleaseCudaKernel(CudaKernelClass* cukernel){
    CudaKernelClass::ReleaseKernel(cukernel);
}

template <class CudaKernelClass>
dim3 FCuda__GetGridSize(CudaKernelClass* /*kernel*/, int intervalSize){
    return CudaKernelClass::GetGridSize(intervalSize);
}

template <class CudaKernelClass>
dim3 FCuda__GetBlockSize(CudaKernelClass* /*kernel*/){
    return CudaKernelClass::GetBlocksSize();
}


/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

#include "FCudaGroupOfCells.hpp"
#include "FCudaGroupAttachedLeaf.hpp"
#include "FCudaGroupOfParticles.hpp"

/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

#include "FCudaEmptyKernel.hpp"
#include "FCudaEmptyCellSymb.hpp"

template void FCuda__bottomPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<int,0,0,int>, FCudaGroupAttachedLeaf<int,0,0,int>, FCudaEmptyKernel<int> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
    unsigned char* containersPtr, std::size_t containersSize,
    FCudaEmptyKernel<int>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__upwardPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<int,0,0,int>, FCudaGroupAttachedLeaf<int,0,0,int>, FCudaEmptyKernel<int> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
int idxLevel, FCudaEmptyKernel<int>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__transferInoutPassCallbackMpi<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<int,0,0,int>, FCudaGroupAttachedLeaf<int,0,0,int>, FCudaEmptyKernel<int> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
    int idxLevel, const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int* safeInteractions, int nbSafeInteractions, FCudaEmptyKernel<int>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__transferInPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<int,0,0,int>, FCudaGroupAttachedLeaf<int,0,0,int>, FCudaEmptyKernel<int> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
    unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
    int idxLevel, FCudaEmptyKernel<int>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__transferInoutPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<int,0,0,int>, FCudaGroupAttachedLeaf<int,0,0,int>, FCudaEmptyKernel<int> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
unsigned char* currentCellsDownPtr,
unsigned char* externalCellsPtr, std::size_t externalCellsSize,
unsigned char* externalCellsUpPtr,
int idxLevel, int mode, const OutOfBlockInteraction* outsideInteractions,
int nbOutsideInteractions,
const int* safeInteractions, int nbSafeInteractions, FCudaEmptyKernel<int>* kernel, hipStream_t currentStream,
                                    const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__downardPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<int,0,0,int>, FCudaGroupAttachedLeaf<int,0,0,int>, FCudaEmptyKernel<int> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
int idxLevel, FCudaEmptyKernel<int>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__directInoutPassCallbackMpi<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<int,0,0,int>, FCudaGroupAttachedLeaf<int,0,0,int>, FCudaEmptyKernel<int> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize,
    const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
const int treeHeight, FCudaEmptyKernel<int>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__directInPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<int,0,0,int>, FCudaGroupAttachedLeaf<int,0,0,int>, FCudaEmptyKernel<int> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    const int treeHeight, FCudaEmptyKernel<int>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__directInoutPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<int,0,0,int>, FCudaGroupAttachedLeaf<int,0,0,int>, FCudaEmptyKernel<int> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
const int     safeOuterInteractions[], const int counterOuterCell,
    const OutOfBlockInteraction* insideInteractions,
    const int     safeInnterInteractions[], const int counterInnerCell,
const int treeHeight, FCudaEmptyKernel<int>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__mergePassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                       FCudaGroupOfParticles<int,0,0,int>, FCudaGroupAttachedLeaf<int,0,0,int>, FCudaEmptyKernel<int> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
    unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    FCudaEmptyKernel<int>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template FCudaEmptyKernel<int>* FCuda__BuildCudaKernel< FCudaEmptyKernel<int> >(void* kernel);
template void FCuda__ReleaseCudaKernel< FCudaEmptyKernel<int> >(FCudaEmptyKernel<int>* cukernel);
template dim3 FCuda__GetGridSize< FCudaEmptyKernel<int> >(FCudaEmptyKernel<int>* kernel, int intervalSize);
template dim3 FCuda__GetBlockSize< FCudaEmptyKernel<int> >(FCudaEmptyKernel<int>* cukernel);

/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

#include "../TestKernel/FCudaTestKernels.hpp"
#include "../TestKernel/FTestCellPOD.hpp"

template void FCuda__bottomPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<float,0, 1, long long int>, FCudaGroupAttachedLeaf<float,0, 1, long long int>, FTestCudaKernels<float> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
unsigned char* containersPtr, std::size_t containersSize,
    FTestCudaKernels<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__upwardPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<float,0, 1, long long int>, FCudaGroupAttachedLeaf<float,0, 1, long long int>, FTestCudaKernels<float> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
int idxLevel, FTestCudaKernels<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__transferInoutPassCallbackMpi<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<float,0, 1, long long int>, FCudaGroupAttachedLeaf<float,0, 1, long long int>, FTestCudaKernels<float> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
    int idxLevel, const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int* safeInteractions, int nbSafeInteractions, FTestCudaKernels<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__transferInPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<float,0, 1, long long int>, FCudaGroupAttachedLeaf<float,0, 1, long long int>, FTestCudaKernels<float> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
    unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
    int idxLevel, FTestCudaKernels<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__transferInoutPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<float,0, 1, long long int>, FCudaGroupAttachedLeaf<float,0, 1, long long int>, FTestCudaKernels<float> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
unsigned char* currentCellsDownPtr,
unsigned char* externalCellsPtr, std::size_t externalCellsSize,
unsigned char* externalCellsUpPtr,
int idxLevel, int mode, const OutOfBlockInteraction* outsideInteractions,
int nbOutsideInteractions,
const int* safeInteractions, int nbSafeInteractions, FTestCudaKernels<float>* kernel, hipStream_t currentStream,
                                    const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__downardPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<float,0, 1, long long int>, FCudaGroupAttachedLeaf<float,0, 1, long long int>, FTestCudaKernels<float> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
int idxLevel, FTestCudaKernels<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__directInoutPassCallbackMpi<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<float,0, 1, long long int>, FCudaGroupAttachedLeaf<float,0, 1, long long int>, FTestCudaKernels<float> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize,
    const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
const int treeHeight, FTestCudaKernels<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__directInPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<float,0, 1, long long int>, FCudaGroupAttachedLeaf<float,0, 1, long long int>, FTestCudaKernels<float> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    const int treeHeight, FTestCudaKernels<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__directInoutPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<float,0, 1, long long int>, FCudaGroupAttachedLeaf<float,0, 1, long long int>, FTestCudaKernels<float> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
const int     safeOuterInteractions[], const int counterOuterCell,
    const OutOfBlockInteraction* insideInteractions,
    const int     safeInnterInteractions[], const int counterInnerCell, const int treeHeight, FTestCudaKernels<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__mergePassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<float,0, 1, long long int>, FCudaGroupAttachedLeaf<float,0, 1, long long int>, FTestCudaKernels<float> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
    unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    FTestCudaKernels<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template FTestCudaKernels<float>* FCuda__BuildCudaKernel<FTestCudaKernels<float>>(void* kernel);
template void FCuda__ReleaseCudaKernel<FTestCudaKernels<float>>(FTestCudaKernels<float>* cukernel);

template dim3 FCuda__GetGridSize< FTestCudaKernels<float> >(FTestCudaKernels<float>* kernel, int intervalSize);
template dim3 FCuda__GetBlockSize< FTestCudaKernels<float> >(FTestCudaKernels<float>* cukernel);




template void FCuda__bottomPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<double,0, 1, long long int>, FCudaGroupAttachedLeaf<double,0, 1, long long int>, FTestCudaKernels<double> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
unsigned char* containersPtr, std::size_t containersSize,
    FTestCudaKernels<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__upwardPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<double,0, 1, long long int>, FCudaGroupAttachedLeaf<double,0, 1, long long int>, FTestCudaKernels<double> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
int idxLevel, FTestCudaKernels<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__transferInoutPassCallbackMpi<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<double,0, 1, long long int>, FCudaGroupAttachedLeaf<double,0, 1, long long int>, FTestCudaKernels<double> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
    int idxLevel, const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int* safeInteractions, int nbSafeInteractions, FTestCudaKernels<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__transferInPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<double,0, 1, long long int>, FCudaGroupAttachedLeaf<double,0, 1, long long int>, FTestCudaKernels<double> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
    unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
    int idxLevel, FTestCudaKernels<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__transferInoutPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<double,0, 1, long long int>, FCudaGroupAttachedLeaf<double,0, 1, long long int>, FTestCudaKernels<double> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
unsigned char* currentCellsDownPtr,
unsigned char* externalCellsPtr, std::size_t externalCellsSize,
unsigned char* externalCellsUpPtr,
int idxLevel, int mode, const OutOfBlockInteraction* outsideInteractions,
int nbOutsideInteractions,
const int* safeInteractions, int nbSafeInteractions, FTestCudaKernels<double>* kernel, hipStream_t currentStream,
                                    const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__downardPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<double,0, 1, long long int>, FCudaGroupAttachedLeaf<double,0, 1, long long int>, FTestCudaKernels<double> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
    int idxLevel, FTestCudaKernels<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__directInoutPassCallbackMpi<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<double,0, 1, long long int>, FCudaGroupAttachedLeaf<double,0, 1, long long int>, FTestCudaKernels<double> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize,
    const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
const int treeHeight, FTestCudaKernels<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__directInPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<double,0, 1, long long int>, FCudaGroupAttachedLeaf<double,0, 1, long long int>, FTestCudaKernels<double> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    const int treeHeight, FTestCudaKernels<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__directInoutPassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<double,0, 1, long long int>, FCudaGroupAttachedLeaf<double,0, 1, long long int>, FTestCudaKernels<double> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
const int     safeOuterInteractions[], const int counterOuterCell,
    const OutOfBlockInteraction* insideInteractions,
    const int     safeInnterInteractions[], const int counterInnerCell, const int treeHeight, FTestCudaKernels<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__mergePassCallback<FTestCellPODCore, FTestCellPODData, FTestCellPODData, FCudaGroupOfCells<FTestCellPODCore, FTestCellPODData, FTestCellPODData>,
                                        FCudaGroupOfParticles<double,0, 1, long long int>, FCudaGroupAttachedLeaf<double,0, 1, long long int>, FTestCudaKernels<double> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
    unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    FTestCudaKernels<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template FTestCudaKernels<double>* FCuda__BuildCudaKernel<FTestCudaKernels<double>>(void* kernel);
template void FCuda__ReleaseCudaKernel<FTestCudaKernels<double>>(FTestCudaKernels<double>* cukernel);

template dim3 FCuda__GetGridSize< FTestCudaKernels<double> >(FTestCudaKernels<double>* kernel, int intervalSize);
template dim3 FCuda__GetBlockSize< FTestCudaKernels<double> >(FTestCudaKernels<double>* cukernel);


/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

#include "../P2P/FCudaP2P.hpp"

template void FCuda__bottomPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FCudaP2P<float> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
unsigned char* containersPtr, std::size_t containersSize,
    FCudaP2P<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__upwardPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FCudaP2P<float> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
    int idxLevel, FCudaP2P<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__transferInoutPassCallbackMpi<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FCudaP2P<float> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
    int idxLevel, const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int* safeInteractions, int nbSafeInteractions, FCudaP2P<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__transferInPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FCudaP2P<float> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
    unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
    int idxLevel, FCudaP2P<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__transferInoutPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FCudaP2P<float> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
unsigned char* currentCellsDownPtr,
unsigned char* externalCellsPtr, std::size_t externalCellsSize,
unsigned char* externalCellsUpPtr,
int idxLevel, int mode, const OutOfBlockInteraction* outsideInteractions,
int nbOutsideInteractions,
const int* safeInteractions, int nbSafeInteractions, FCudaP2P<float>* kernel, hipStream_t currentStream,
                                    const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__downardPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FCudaP2P<float> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
int idxLevel, FCudaP2P<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__directInoutPassCallbackMpi<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FCudaP2P<float> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize,
    const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
const int treeHeight, FCudaP2P<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__directInPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FCudaP2P<float> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    const int treeHeight, FCudaP2P<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__directInoutPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FCudaP2P<float> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
const int     safeOuterInteractions[], const int counterOuterCell,
    const OutOfBlockInteraction* insideInteractions,
    const int     safeInnterInteractions[], const int counterInnerCell, const int treeHeight, FCudaP2P<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__mergePassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FCudaP2P<float> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
    unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    FCudaP2P<float>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template FCudaP2P<float>* FCuda__BuildCudaKernel<FCudaP2P<float>>(void* kernel);
template void FCuda__ReleaseCudaKernel<FCudaP2P<float>>(FCudaP2P<float>* cukernel);

template dim3 FCuda__GetGridSize< FCudaP2P<float> >(FCudaP2P<float>* kernel, int intervalSize);
template dim3 FCuda__GetBlockSize< FCudaP2P<float> >(FCudaP2P<float>* cukernel);




template void FCuda__bottomPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FCudaP2P<double> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
unsigned char* containersPtr, std::size_t containersSize,
    FCudaP2P<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__upwardPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FCudaP2P<double> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
int idxLevel, FCudaP2P<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__transferInoutPassCallbackMpi<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FCudaP2P<double> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
    int idxLevel, const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int* safeInteractions, int nbSafeInteractions, FCudaP2P<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__transferInPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FCudaP2P<double> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
    unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
    int idxLevel, FCudaP2P<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__transferInoutPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FCudaP2P<double> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
unsigned char* currentCellsDownPtr,
unsigned char* externalCellsPtr, std::size_t externalCellsSize,
unsigned char* externalCellsUpPtr,
int idxLevel, int mode, const OutOfBlockInteraction* outsideInteractions,
int nbOutsideInteractions,
const int* safeInteractions, int nbSafeInteractions, FCudaP2P<double>* kernel, hipStream_t currentStream,
                                    const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__downardPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FCudaP2P<double> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
int idxLevel, FCudaP2P<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__directInoutPassCallbackMpi<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FCudaP2P<double> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize,
    const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
const int treeHeight, FCudaP2P<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__directInPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FCudaP2P<double> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    const int treeHeight, FCudaP2P<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__directInoutPassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FCudaP2P<double> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
const int     safeOuterInteractions[], const int counterOuterCell,
    const OutOfBlockInteraction* insideInteractions,
    const int     safeInnterInteractions[], const int counterInnerCell, const int treeHeight, FCudaP2P<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__mergePassCallback<FCudaEmptyCellSymb, int, int, FCudaGroupOfCells<FCudaEmptyCellSymb, int, int>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FCudaP2P<double> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
    unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    FCudaP2P<double>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template FCudaP2P<double>* FCuda__BuildCudaKernel<FCudaP2P<double>>(void* kernel);
template void FCuda__ReleaseCudaKernel<FCudaP2P<double>>(FCudaP2P<double>* cukernel);

template dim3 FCuda__GetGridSize< FCudaP2P<double> >(FCudaP2P<double>* kernel, int intervalSize);
template dim3 FCuda__GetBlockSize< FCudaP2P<double> >(FCudaP2P<double>* cukernel);



/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

#include "../Uniform/FUnifCuda.hpp"

template void FCuda__bottomPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,5> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
unsigned char* containersPtr, std::size_t containersSize,
    FUnifCuda<float,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__upwardPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,5> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
    int idxLevel, FUnifCuda<float,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__transferInoutPassCallbackMpi<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,5> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
    int idxLevel, const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int* safeInteractions, int nbSafeInteractions, FUnifCuda<float,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__transferInPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,5> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
    unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
    int idxLevel, FUnifCuda<float,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__transferInoutPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,5> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
unsigned char* currentCellsDownPtr,
unsigned char* externalCellsPtr, std::size_t externalCellsSize,
unsigned char* externalCellsUpPtr,
int idxLevel, int mode, const OutOfBlockInteraction* outsideInteractions,
int nbOutsideInteractions,
const int* safeInteractions, int nbSafeInteractions, FUnifCuda<float,5>* kernel, hipStream_t currentStream,
                                    const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__downardPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,5> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
int idxLevel, FUnifCuda<float,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__directInoutPassCallbackMpi<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,5> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize,
    const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
const int treeHeight, FUnifCuda<float,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__directInPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,5> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    const int treeHeight, FUnifCuda<float,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__directInoutPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,5> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
const int     safeOuterInteractions[], const int counterOuterCell,
    const OutOfBlockInteraction* insideInteractions,
    const int     safeInnterInteractions[], const int counterInnerCell, const int treeHeight, FUnifCuda<float,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__mergePassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,5>,FCudaUnifCellPODLocal<float,5>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,5> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
    unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    FUnifCuda<float,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template FUnifCuda<float,5>* FCuda__BuildCudaKernel<FUnifCuda<float,5>>(void* kernel);
template void FCuda__ReleaseCudaKernel<FUnifCuda<float,5>>(FUnifCuda<float,5>* cukernel);

template dim3 FCuda__GetGridSize< FUnifCuda<float,5> >(FUnifCuda<float,5>* kernel, int intervalSize);
template dim3 FCuda__GetBlockSize< FUnifCuda<float,5> >(FUnifCuda<float,5>* cukernel);

template void FUnifCudaFillObject(void* cudaKernel, const FUnifCudaSharedData<double,5>& hostData);



template void FCuda__bottomPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,5> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
unsigned char* containersPtr, std::size_t containersSize,
    FUnifCuda<double,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__upwardPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,5> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
int idxLevel, FUnifCuda<double,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__transferInoutPassCallbackMpi<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,5> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
    int idxLevel, const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int* safeInteractions, int nbSafeInteractions, FUnifCuda<double,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__transferInPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,5> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
    unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
    int idxLevel, FUnifCuda<double,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__transferInoutPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,5> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
unsigned char* currentCellsDownPtr,
unsigned char* externalCellsPtr, std::size_t externalCellsSize,
unsigned char* externalCellsUpPtr,
int idxLevel, int mode, const OutOfBlockInteraction* outsideInteractions,
int nbOutsideInteractions,
const int* safeInteractions, int nbSafeInteractions, FUnifCuda<double,5>* kernel, hipStream_t currentStream,
                                    const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__downardPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,5> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
int idxLevel, FUnifCuda<double,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__directInoutPassCallbackMpi<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,5> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize,
    const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
const int treeHeight, FUnifCuda<double,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__directInPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,5> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    const int treeHeight, FUnifCuda<double,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__directInoutPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,5> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
const int     safeOuterInteractions[], const int counterOuterCell,
    const OutOfBlockInteraction* insideInteractions,
    const int     safeInnterInteractions[], const int counterInnerCell, const int treeHeight, FUnifCuda<double,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__mergePassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,5>,FCudaUnifCellPODLocal<double,5>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,5> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
    unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    FUnifCuda<double,5>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template FUnifCuda<double,5>* FCuda__BuildCudaKernel<FUnifCuda<double,5>>(void* kernel);
template void FCuda__ReleaseCudaKernel<FUnifCuda<double,5>>(FUnifCuda<double,5>* cukernel);

template dim3 FCuda__GetGridSize< FUnifCuda<double,5> >(FUnifCuda<double,5>* kernel, int intervalSize);
template dim3 FCuda__GetBlockSize< FUnifCuda<double,5> >(FUnifCuda<double,5>* cukernel);

template void FUnifCudaFillObject(void* cudaKernel, const FUnifCudaSharedData<float,5>& hostData);




template void FCuda__bottomPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,7> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
unsigned char* containersPtr, std::size_t containersSize,
    FUnifCuda<float,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__upwardPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,7> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
    int idxLevel, FUnifCuda<float,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__transferInoutPassCallbackMpi<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,7> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
    int idxLevel, const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int* safeInteractions, int nbSafeInteractions, FUnifCuda<float,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__transferInPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,7> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
    unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
    int idxLevel, FUnifCuda<float,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__transferInoutPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,7> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
unsigned char* currentCellsDownPtr,
unsigned char* externalCellsPtr, std::size_t externalCellsSize,
unsigned char* externalCellsUpPtr,
int idxLevel, int mode, const OutOfBlockInteraction* outsideInteractions,
int nbOutsideInteractions,
const int* safeInteractions, int nbSafeInteractions, FUnifCuda<float,7>* kernel, hipStream_t currentStream,
                                    const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__downardPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,7> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
int idxLevel, FUnifCuda<float,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__directInoutPassCallbackMpi<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,7> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize,
    const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
const int treeHeight, FUnifCuda<float,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__directInPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,7> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    const int treeHeight, FUnifCuda<float,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__directInoutPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,7> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
const int     safeOuterInteractions[], const int counterOuterCell,
    const OutOfBlockInteraction* insideInteractions,
    const int     safeInnterInteractions[], const int counterInnerCell, const int treeHeight, FUnifCuda<float,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__mergePassCallback<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<float,7>,FCudaUnifCellPODLocal<float,7>>,
                                        FCudaGroupOfParticles<float,1, 4, float>, FCudaGroupAttachedLeaf<float,1, 4, float>, FUnifCuda<float,7> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
    unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    FUnifCuda<float,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template FUnifCuda<float,7>* FCuda__BuildCudaKernel<FUnifCuda<float,7>>(void* kernel);
template void FCuda__ReleaseCudaKernel<FUnifCuda<float,7>>(FUnifCuda<float,7>* cukernel);

template dim3 FCuda__GetGridSize< FUnifCuda<float,7> >(FUnifCuda<float,7>* kernel, int intervalSize);
template dim3 FCuda__GetBlockSize< FUnifCuda<float,7> >(FUnifCuda<float,7>* cukernel);

template void FUnifCudaFillObject(void* cudaKernel, const FUnifCudaSharedData<double,7>& hostData);



template void FCuda__bottomPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,7> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsUpPtr,
unsigned char* containersPtr, std::size_t containersSize,
    FUnifCuda<double,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__upwardPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,7> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsUpPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsUpPtr,
int idxLevel, FUnifCuda<double,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__transferInoutPassCallbackMpi<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,7> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* externalCellsPtr, std::size_t externalCellsSize, unsigned char* externalCellsUpPtr,
    int idxLevel, const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int* safeInteractions, int nbSafeInteractions, FUnifCuda<double,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__transferInPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,7> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
    unsigned char* currentCellsUpPtr, unsigned char* currentCellsDownPtr,
    int idxLevel, FUnifCuda<double,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__transferInoutPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,7> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize,
unsigned char* currentCellsDownPtr,
unsigned char* externalCellsPtr, std::size_t externalCellsSize,
unsigned char* externalCellsUpPtr,
int idxLevel, int mode, const OutOfBlockInteraction* outsideInteractions,
int nbOutsideInteractions,
const int* safeInteractions, int nbSafeInteractions, FUnifCuda<double,7>* kernel, hipStream_t currentStream,
                                    const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__downardPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,7> >
    (unsigned char* currentCellsPtr, std::size_t currentCellsSize, unsigned char* currentCellsDownPtr,
    unsigned char* childCellsPtr, std::size_t childCellsSize, unsigned char* childCellsDownPtr,
int idxLevel, FUnifCuda<double,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#ifdef SCALFMM_USE_MPI
template void FCuda__directInoutPassCallbackMpi<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,7> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize,
    const OutOfBlockInteraction* outsideInteractions,
    int nbOutsideInteractions, const int safeOuterInteractions[], const int counterOuterCell,
const int treeHeight, FUnifCuda<double,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);
#endif
template void FCuda__directInPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,7> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    const int treeHeight, FUnifCuda<double,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__directInoutPassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,7> >
    (unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    unsigned char* externalContainersPtr, std::size_t externalContainersSize, unsigned char* externalContainersDownPtr,
const OutOfBlockInteraction* outsideInteractions, int nbOutsideInteractions,
const int     safeOuterInteractions[], const int counterOuterCell,
    const OutOfBlockInteraction* insideInteractions,
    const int     safeInnterInteractions[], const int counterInnerCell, const int treeHeight, FUnifCuda<double,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template void FCuda__mergePassCallback<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>, FCudaGroupOfCells<FBasicCellPOD, FCudaUnifCellPODPole<double,7>,FCudaUnifCellPODLocal<double,7>>,
                                        FCudaGroupOfParticles<double,1, 4, double>, FCudaGroupAttachedLeaf<double,1, 4, double>, FUnifCuda<double,7> >
    (unsigned char* leafCellsPtr, std::size_t leafCellsSize, unsigned char* leafCellsDownPtr,
    unsigned char* containersPtr, std::size_t containersSize, unsigned char* containersDownPtr,
    FUnifCuda<double,7>* kernel, hipStream_t currentStream,
                                        const dim3 inGridSize, const dim3 inBlocksSize);

template FUnifCuda<double,7>* FCuda__BuildCudaKernel<FUnifCuda<double,7>>(void* kernel);
template void FCuda__ReleaseCudaKernel<FUnifCuda<double,7>>(FUnifCuda<double,7>* cukernel);

template dim3 FCuda__GetGridSize< FUnifCuda<double,7> >(FUnifCuda<double,7>* kernel, int intervalSize);
template dim3 FCuda__GetBlockSize< FUnifCuda<double,7> >(FUnifCuda<double,7>* cukernel);

template void FUnifCudaFillObject(void* cudaKernel, const FUnifCudaSharedData<float,7>& hostData);
